
#include <hip/hip_runtime.h>
#include <iostream>
#include <algorithm>
#include <stdio.h>
#include <omp.h>
#include <string>

#define BLOCK_SIZE 1024

void Check_CUDA_Error(const char *message){
  hipError_t error = hipGetLastError();
  if(error!=hipSuccess) {
    fprintf(stderr,"ERROR: %s: %s\n", message, hipGetErrorString(error) );
    exit(-1);
  }
}

__global__ void reduction(double* sum, const double* a, long N){
  __shared__ double smem[BLOCK_SIZE];
  int idx = (blockIdx.x) * blockDim.x + threadIdx.x;

  if (idx < N) 
    smem[threadIdx.x] = a[idx];
  else 
    smem[threadIdx.x] = 0;

  __syncthreads();
  if (threadIdx.x < 512) 
    smem[threadIdx.x] += smem[threadIdx.x + 512];
  __syncthreads();
  if (threadIdx.x < 256) 
    smem[threadIdx.x] += smem[threadIdx.x + 256];
  __syncthreads();
  if (threadIdx.x < 128) 
    smem[threadIdx.x] += smem[threadIdx.x + 128];
  __syncthreads();
  if (threadIdx.x <  64) 
    smem[threadIdx.x] += smem[threadIdx.x +  64];
  __syncthreads();
  if (threadIdx.x <  32) {
    smem[threadIdx.x] += smem[threadIdx.x +  32];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +  16];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +   8];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +   4];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +   2];
    __syncwarp();
    if (threadIdx.x == 0)
      sum[blockIdx.x] = smem[0] + smem[1];
  }
}

__global__ void matrixVecMult(double* sum, const double* A, const double* b, long N){
  __shared__ double smem[BLOCK_SIZE];
  int idx = (blockIdx.x) * blockDim.x + threadIdx.x;

  if (idx < N) 
    smem[threadIdx.x] = A[idx]*b[idx];
  else 
    smem[threadIdx.x] = 0;

  __syncthreads();
  if (threadIdx.x < 512) 
    smem[threadIdx.x] += smem[threadIdx.x + 512];
  __syncthreads();
  if (threadIdx.x < 256) 
    smem[threadIdx.x] += smem[threadIdx.x + 256];
  __syncthreads();
  if (threadIdx.x < 128) 
    smem[threadIdx.x] += smem[threadIdx.x + 128];
  __syncthreads();
  if (threadIdx.x <  64) 
    smem[threadIdx.x] += smem[threadIdx.x +  64];
  __syncthreads();
  if (threadIdx.x <  32) {
    smem[threadIdx.x] += smem[threadIdx.x +  32];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +  16];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +   8];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +   4];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +   2];
    __syncwarp();
    
    if (threadIdx.x == 0) 
      sum[blockIdx.x] = smem[0] + smem[1];
  }
}

void matrixVecMult_CPU(double* C, const double* A, const double* B, long N) {
  for (long i = 0; i < N; i++) {
    double sum = 0;
    #pragma omp parallel for schedule(static) reduction(+:sum)
    for (long j = 0; j < N; j++) {
      sum += A[i*N+j]*B[j];
    }
    C[i] = sum;
  }
}

int main() {
    long N;
    int exp;
    std::cout << "N = 2^ ";
    std::cin >> exp;
    N = (1UL<<exp);

    double *x = (double*)malloc(sizeof(double) * N);
    #pragma omp parallel for schedule(static)
    for (long i = 0; i < N; i++) 
      x[i] = drand48();

    double *A = (double*)malloc(sizeof(double) * N*N);
    #pragma omp parallel for schedule(static)
    for (long i = 0; i < N*N; i++) 
      A[i] = drand48();

    double *sum_ref, *sum;
    hipHostMalloc((void**)&sum_ref, N * sizeof(double), hipHostMallocDefault);
    hipHostMalloc((void**)&sum, N * sizeof(double), hipHostMallocDefault);
  
    double tt = omp_get_wtime();
    matrixVecMult_CPU(sum_ref, A, x, N);

    printf("CPU Bandwidth = %f GB/s\n", N*N*sizeof(double) / (omp_get_wtime()-tt)/1e9);

    double *x_d, *A_d, *z_d;
    hipMalloc(&x_d, N*sizeof(double));
    hipMalloc(&A_d, N*N*sizeof(double));
    
    long N_work = 1;
   
    for (long i = (N+BLOCK_SIZE-1)/(BLOCK_SIZE); i > 1; i = (i+BLOCK_SIZE-1)/(BLOCK_SIZE)) 
      N_work += i;
    
    hipMalloc(&z_d, N_work*sizeof(double)); // extra memory buffer for reduction across thread-blocks

    hipMemcpyAsync(x_d, x, N*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(A_d, A, N*N*sizeof(double), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    tt = omp_get_wtime();

    for (long i = 0; i < N; i++) {
      double* sum_d = z_d;
      long Nb = (N+BLOCK_SIZE-1) / (BLOCK_SIZE);
      
      matrixVecMult<<<Nb, BLOCK_SIZE>>>(sum_d, A_d+i*N, x_d, N);
      
      while (Nb > 1) {
        long Nx = Nb;
        Nb = (Nb+BLOCK_SIZE-1)/(BLOCK_SIZE);
        reduction<<<Nb,BLOCK_SIZE>>>(sum_d + Nx, sum_d, Nx);
        sum_d += Nx;
      }
      
      hipMemcpyAsync(&sum[i], sum_d, 1*sizeof(double), hipMemcpyDeviceToHost);
      hipDeviceSynchronize();
    }

    printf("GPU Bandwidth = %f GB/s\n", N*N*sizeof(double) / (omp_get_wtime()-tt)/1e9);
    
    double error = 0;
    #pragma omp parallel for reduction(+:error)
    for (long i = 0; i < N; i++)
      error = error + fabs(sum[i] - sum_ref[i]);
    
    printf("Error = %f\n", error);
    
    hipFree(x_d);
    hipFree(z_d);
    hipFree(A_d); 
    hipHostFree(x);
    hipHostFree(A);

    return 0;
}